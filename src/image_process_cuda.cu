#include "hip/hip_runtime.h"
#include "../includes/image_process_cuda.h"

#include <iostream>
#include <unordered_map>
#include <vector>

#include <opencv2/opencv.hpp>

// __global__ void _getImageChunk(uchar* originalData, uchar* pixelData,
//     const int rows, const int cols) {
//   const int kIntensityLevels = 20;
//   const int kRadius = 5;
//   const int kGridIdx = (blockIdx.y * gridDim.x) + blockIdx.x;
//   const int kBlock = kGridIdx * blockDim.x * blockDim.y;
//   const int kThreadId = threadIdx.y * blockDim.x + threadIdx.x;
//   const int kPixelIndex = kBlock + kThreadId;

//   if (kPixelIndex < rows * cols) {
//     const int kRow = floorf(kPixelIndex / cols);
//     const int kColumn = kPixelIndex % cols;

//     int maximumIntensity = -1;

//     // 255 because there won't be more than 255 intensities
//     int colorTotalsR[255] = {0};
//     int colorTotalsG[255] = {0};
//     int colorTotalsB[255] = {0};
//     int intensityCount[255] = {0};

//     for (int ii = max(0, kRow - kRadius);
//         ii < min(rows, kRow + kRadius); ++ii) {
//       for (int jj = max(0, kColumn - kRadius);
//             jj < min(cols, kColumn + kRadius); ++jj) {
//         const int kRow2 = floorf(ii / cols);
//         const int kColumn2 = ii % cols;
          
//         const double kR = originalData[kRow2 * cols + kColumn2 + 2];
//         const double kG = originalData[kRow2 * cols + kColumn2 + 1];
//         const double kB = originalData[kRow2 * cols + kColumn2];

//         int intensity = (((kR + kG + kB) / 3) * kIntensityLevels) / 255.0f;
//         if (intensity > 255) {
//           intensity = 255;
//         }

//         ++intensityCount[intensity];

//         if (maximumIntensity == -1 || maximumIntensity < intensity) {
//           maximumIntensity = intensity;
//         }

//         colorTotalsR[intensity] += kR;
//         colorTotalsG[intensity] += kR;
//         colorTotalsB[intensity] += kR;
//       }
//     }

//     const int kRFinal =
//         colorTotalsR[maximumIntensity] / intensityCount[maximumIntensity];
//     const int kGFinal =
//         colorTotalsG[maximumIntensity] / intensityCount[maximumIntensity];
//     const int kBFinal =
//         colorTotalsB[maximumIntensity] / intensityCount[maximumIntensity];

//     pixelData[kRow * 3 * cols + kColumn * 3 + 2] = kRFinal;
//     pixelData[kRow * 3 * cols + kColumn * 3 + 1] = kGFinal;
//     pixelData[kRow * 3 * cols + kColumn * 3] = kBFinal;
//   }
// }

cv::Mat getProcessedImageParallelCUDA(const cv::Mat& image) {
  // const int kSize = image.rows * image.cols;
  // std::vector<uchar> pixelData(kSize * 3); // because 3 channels
  // cv::cuda::GpuMat newGpuImage(image.rows, image.cols, CV_8UC3,
  //     pixelData.data());
  // cv::cuda::GpuMat gpuImage(image);
  // const int kThreadAmountPerBlock = 32;
  // const int kBlockAmount =
  //     (int)ceil((image.rows * image.cols) / kThreadAmountPerBlock);
  // _getImageChunk<<<kBlockAmount, kThreadAmountPerBlock>>>(gpuImage.data,
  //     newGpuImage.data, image.rows, image.cols);

  cv::Mat outputImage(image.rows, image.cols, CV_8UC3);
  // newGpuImage.download(outputImage);
  return outputImage;
}
