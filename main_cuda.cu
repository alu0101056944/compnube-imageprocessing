/**
 * Author: Marcos Barrios
 * Since: 22/04/2024
 * Description: Print execution time of cuda parallel image processing oil
 *    painting algorithm.
*/

#include <iostream>
#include <string>
#include <chrono>
#include <filesystem>
#include <sys/timeb.h>
#include <sys/types.h>

#include <opencv2/opencv.hpp>

namespace fs = std::filesystem;

// #include "includes/image_process_cuda.h"

// void printExecutionTime(const cv::Mat& image) {
//   const int kAmountOfIterations = 5;

//   struct _timeb timeInit;
//   struct _timeb timeEnd;
//   _ftime(&timeInit);
//   for (size_t i = 0; i < kAmountOfIterations; ++i) {
//     const cv::Mat processedImage = getProcessedImageParallelCUDA(image);
//   }
//   _ftime(&timeEnd);

//   time_t kEjecutionTime = timeEnd.time - timeInit.time +
//       (timeEnd.millitm - timeInit.millitm) / 1.0e3;
//   std::cout << kEjecutionTime / kAmountOfIterations;
//   std::cout << " seconds. (Execution time)" << std::endl;
// }

// void writeImage(const cv::Mat& image, const fs::path& path) {
//   std::cout << "Writing image, please wait..." << std::endl;
//   const cv::Mat outputImage = getProcessedImageParallelCUDA(image);
//   const std::string kOutputPath = (path.parent_path() / path.stem())
//       .string() + "_processed" + path.extension().string();
//   cv::imwrite(kOutputPath, outputImage);
// }

int main(int argc, char** argv) {
  std::cout << "something here" << std::endl;
  if (argc < 2) {
    std::cout << "Usage: <path to image> (expected at least one argument).";
    std::cout << std::endl;
    return -1;
  }

  std::cout << "something here" << std::endl;

  const std::string kFilePath = argv[1];
  fs::path inputPath(kFilePath);
  if (!fs::exists(inputPath) ||
      !fs::is_regular_file(inputPath)) {
    std::cout << "Invalid file path." << std::endl;
    return -1;
  }

  cv::Mat image = cv::imread(kFilePath);
  // if (image.empty()) {
  //   std::cerr << "Error: Unable to load image." << std::endl;
  //   return -1;
  // }

  // printExecutionTime(image);
  // writeImage(image, inputPath);

  return 0;
}
